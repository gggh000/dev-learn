#include "hip/hip_runtime.h"
/**
 */

/**
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

__global__ void add( int a, int b, int * c) {
	*c  = a + b;
}

int main ( void ) {
	int c;
	int * dev_c;
	hipMalloc( ( void ** ) & dev_c, sizeof(int) );
	add <<<1,1>>> (2,7, dev_c);
	hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	printf( " 2 + 7 = %d\n", c);
}
