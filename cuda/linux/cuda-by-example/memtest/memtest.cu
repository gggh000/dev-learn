#include "hip/hip_runtime.h"
/**
This example demonstrates what happens if the vector sum addition exceeds the threads per block.
The N is assigned the threads / block size right from prop. As long as the vector size is withing this
size, the sum will succeed. However, if it exceeds, the sum will fail and c[] array will return with 
garbage. It will not return the partial data, the part of the vector that fits within the threads per block.
Rather, whole vector sum return data will be invalid. To prove this, increase the N such that N > threads/block.

 */

/**
 */

// System includes

#include <stdio.h>
#include <assert.h>

// CUDA runtime

#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define N 1024*1024*1024
#define DEBUG 0

__global__ void memtrf(char * a) {
	//int tid = threadIdx.x + blockIdx.x * blockDim.x;
}
int main ( void ) {
	char *dev_a;
	int blockDim;
	hipEvent_t start, stop;
	//int errors;
	char * a;

        hipDeviceProp_t prop;
        int count, i;
        hipGetDeviceCount ( &count);
	float elapsedTime;

        for (i = 0 ; i < count ; i ++ )
                hipGetDeviceProperties ( &prop, i);

	blockDim = prop.maxThreadsPerBlock;
	printf("Max threads per block for device 0: %d", blockDim);

	//int a[N];
	a = (char*) malloc(sizeof(char) * N);
	printf("\nAllocated memory on the host ok: 0x%08x", N);
	hipMalloc( (void**) &dev_a, N * sizeof(char));
	printf("\nAllocated memory on the GPU ok: 0x%08x", N);


	hipEventCreate( &start);
	hipEventCreate( &stop);
	hipEventRecord( start, 0);
	hipMemcpy(dev_a, a, N * sizeof(char), hipMemcpyHostToDevice);
	hipEventRecord( stop, 0);
	hipEventElapsedTime( &elapsedTime, start, stop);
	printf("\nTime taken: %3.1f ms\n", elapsedTime);
	
	//memtrf <<<(N + blockDim - 1) / blockDim, blockDim>>>(dev_a);
	//hipMemcpy(c, dev_c, N * sizeof(char), hipMemcpyDeviceToHost);

	/*
	for (int i = 0; i < N; i++) {
		if (c[i] != 6) {
			//printf("\n0x%x did not add correctly: %d", i, c[i]);
			errors ++;
			//continue;
		}
		//printf("\n%d. GPU address: a/b/c: 0x%08x, 0x%08x, 0x%08x, host addr: 0x%0x", i, a[i], b[i], c[i], &c[i]);
	}
	*/

	printf("\nsize of int, long int: %d, %d", sizeof(char), sizeof(long int));

	printf("\nPress a key to release the cuda memory...");
	getchar();
	hipFree(dev_a);

	//printf("\nNo. of errors in vector sum: %d.", errors);
	printf("\n");
}
