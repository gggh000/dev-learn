
#include <hip/hip_runtime.h>
#include <stdio.h>

// 1. if N is set to up to 1024, then sum is OK.
// 2. Set N past the 1024 which is past No. of threads per blocks, and then all iterations of sum results in 
// even the ones within the block.

// 3. To circumvent the problem described in 2. above, since if N goes past No. of threads per block, we need multiple block launch.
// The trick is describe in p65 to use formula (N+127) / 128 for blocknumbers so that when block number starts from 1, it is 
// (1+127) / 128.

#define N 4096

__global__ void add( int * a, int * b, int * c ) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	while (tid < N) {
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

int main (void) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	hipMalloc( (void**)&dev_a, N * sizeof(int));
	hipMalloc( (void**)&dev_b, N * sizeof(int));
	hipMalloc( (void**)&dev_c, N * sizeof(int));

	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i * i;
	}

	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	add<<<(N+127)/128, 128>>> (dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i+=100) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}
