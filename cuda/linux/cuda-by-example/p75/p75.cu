// shared memory concept. 

#include <stdio.h>
#include <hip/hip_runtime.h>


#define imin(a,b) (a<b?a:b)

const int N = 33 * 1024;
const int threadsPerBlock = 256;

__global__ void dot(float * a, float * b, float * c) {

    // this variable will be repeated and will have same value on every blocks.

    __shared__ float cache[threadsPerBlock];

    int tid = threadIdx.x * blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while (tid < N)  {

        // temp will accumulate the product of a and b at every at every grid.

        temp += a[tid] * b[tid] ;
        tid += blockDim.x * gridDim.x;
    }

    // holds product of a and b at every grid. 

    cache[cacheIndex] = temp; 

    // sync threads in this block.

    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2 because of following code.

    int i = blockDim.x/2; 

    while (i != 0 ) {
        if (cacheIndex < i) 
            cache[cacheIndex] += cache[cacheIndex + i];

        __syncthreads();
        i /= 2 ;
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}

int main( void ) {
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;

    const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1 ) / threadsPerBlock) ;

    // allocate memory on the cpu side.

    a = (float*)malloc( N * sizeof(float));
    b = (float*)malloc( N * sizeof(float));
    partial_c = (float*)malloc( blocksPerGrid * sizeof(float));

    // allocate memory on the gpu side.

    hipMalloc((void**) &dev_a, N * sizeof(float));
    hipMalloc((void**) &dev_b, N * sizeof(float));
    hipMalloc((void**) &dev_partial_c, blocksPerGrid * sizeof(float));

    // file in the host memory with data.
    
    for (int i = 0; i < N; i++ ) {
        a[i] = i;
        b[i] = i * 2;
    }

    // copy the arrays a and b to the gpu.

    hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

    dot<<< blocksPerGrid, threadsPerBlock>>> ( dev_a, dev_b, dev_partial_c);

    hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);

    c = 0;    
    for (int i = 0; i < blocksPerGrid; i++)
        c += partial_c[i];

    #define sum_squares(x) (x*(x+1)*(2*x+1)/6)

    printf("Does GPU value %.6g = %.6g?\n", c, 2 * sum_squares((float) (N-1)));

    // free memory on gpu side.

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_partial_c);

    // free on cpu side.

    free(a);
    free(b);
    free(partial_c);
}
