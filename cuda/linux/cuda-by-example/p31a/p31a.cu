#include "hip/hip_runtime.h"
/**
This example demonstrates what happens if the vector sum addition exceeds the threads per block.
The N is assigned the threads / block size right from prop. As long as the vector size is withing this
size, the sum will succeed. However, if it exceeds, the sum will fail and c[] array will return with 
garbage. It will not return the partial data, the part of the vector that fits within the threads per block.
Rather, whole vector sum return data will be invalid. To prove this, increase the N such that N > threads/block.

 */

/**
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

//#define N 1025

__global__ void add( long int * a, long int * b, long int * c, int N) {
	int tid = threadIdx.x;
	if ( tid < N )
		c[tid] = a[tid] + b[tid];
		c[tid] = (long int)&c[tid];
}
int main ( void ) {
	long int *dev_a, *dev_b, *dev_c;
	int errors, N;

        hipDeviceProp_t prop;
        int count, i;
        hipGetDeviceCount ( &count);

        for (i = 0 ; i < count ; i ++ )
                hipGetDeviceProperties ( &prop, i);

	N = prop.maxThreadsPerBlock;
	N = 10;
	printf("Max threads per block for device 0: %d", N);
	

	int a[N], b[N], c[N];

	hipMalloc( (void**) &dev_a, N * sizeof(int));
	hipMalloc( (void**) &dev_b, N * sizeof(int));
	hipMalloc( (void**) &dev_c, N * sizeof(int));

	errors = 0;

	for (int i = 0; i < N; i++) {
		a[i] = 2;
		b[i] = 4;
	}

	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	add <<<1, N>>>(dev_a, dev_b, dev_c, N);

	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++) {
		if (c[i] != 6) {
			//printf("\n0x%x did not add correctly: %d", i, c[i]);
			errors ++;
			//continue;
		}
		printf("\n%d. GPU address: 0x%0x, host addr: 0x%0x", i, c[i], &c[i]);
	}

	printf("\nsize of int, long int: %d, %d", sizeof(int), sizeof(long int));
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	printf("\nNo. of errors in vector sum: %d.", errors);
	printf("\n");
}
