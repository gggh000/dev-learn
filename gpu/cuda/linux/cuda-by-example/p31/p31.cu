/**
 */

/**
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>

int main ( void ) {
	hipDeviceProp_t prop;
	int count, i;
	hipGetDeviceCount ( &count);
  
    printf("No. of cuda devices: %u.\n", count);

	for (i = 0 ; i < count ; i ++ ) {	
		hipGetDeviceProperties ( &prop, i);
		printf("\n\n====DEVICE %d=====", i);
		printf("\nDevice name: 		%s", prop.name);
		printf("\nTotal global mem: 	0x%0x", prop.totalGlobalMem);
		printf("\nwarpSize: 		0x%0x", prop.warpSize);
		printf("\nmaxThreads/Block: 	0x%0x", prop.maxThreadsPerBlock);
		printf("\nmaxThreads/DIM: 	0x%0x", prop.maxThreadsDim);
		printf("\nmaxGridSize: 		0x%0x", prop.maxGridSize);
		printf("\ntotal const mem: 	0x%0x", prop.totalConstMem);
		printf("\ncompute cap: 		%d.%d", prop.major, prop.minor);
		printf("\nmultiprocessors: 	0x%0x", prop.multiProcessorCount);
		printf("\ncanMapHostMem: 	0x%0x", prop.canMapHostMemory);
		printf("\ncomputeMode: 		0x%0x", prop.computeMode);
		printf("\nconcurrentKernels: 	0x%0x", prop.concurrentKernels);
		printf("\nmaxThreadsPerMultiProcessor: 	0x%0x", prop.maxThreadsPerMultiProcessor);
		printf("\nsharedMemPerMultiprocessor: 	0x%0x", prop.sharedMemPerMultiprocessor);
		printf("\nregsPerMultiprocessor: 	0x%0x", prop.regsPerMultiprocessor);
		printf("\nregsPerBlock: 	0x%0x", prop.regsPerBlock);
        printf("\nmultiProcessorCount: 0x%5x", prop.multiProcessorCount);
	}

	printf("\n");
}
