/* source of this code:
https://docs.nvidia.com/cuda/cublas/index.html#abstract 
Example 1. Application Using C and cuBLAS: 1-based indexing
----------------------------------------------------------- */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define M 6
#define N 5
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

static __inline__ void modify (hipblasHandle_t handle, float *m, int ldm, int n, int p, int q, float alpha, float beta){
    hipblasSscal (handle, n-q+1, &alpha, &m[IDX2F(p,q,ldm)], ldm);
    hipblasSscal (handle, ldm-p+1, &beta, &m[IDX2F(p,q,ldm)], 1);
}

int main (void){
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int i, j;
    float* devPtrA;
    float* a = 0;

    // allocate 2D matrix continuius main memory of M*N size * float.

    a = (float *)malloc (M * N * sizeof (*a));
    if (!a) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }

    // assign values to array.
    
    for (j = 1; j <= N; j++) {
        for (i = 1; i <= M; i++) {
            a[IDX2F(i,j,M)] = (float)((i-1) * N + j);
        }
    }

    // allocate 2D matrix in gpu memory M*N pointed by devPtrA.

    cudaStat = hipMalloc ((void**)&devPtrA, M*N*sizeof(*a));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }

    // cublas initialization.

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    // set matrix, copy from cpu to gpu memory.

    stat = hipblasSetMatrix (M, N, sizeof(*a), a, M, devPtrA, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    // scale the matrix.

    modify (handle, devPtrA, M, N, 2, 3, 16.0f, 12.0f);

    // copies matrix from gpu to cpu memory.

    stat = hipblasGetMatrix (M, N, sizeof(*a), devPtrA, M, a, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    // clean up.

    hipFree (devPtrA);
    hipblasDestroy(handle);
    for (j = 1; j <= N; j++) {
        for (i = 1; i <= M; i++) {
            printf ("%7.0f", a[IDX2F(i,j,M)]);
        }
        printf ("\n");
    }
    free(a);
    return EXIT_SUCCESS;
}
