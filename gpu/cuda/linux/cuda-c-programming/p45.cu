#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) 
        C[i] = A[i] + B[i];
}

void initialData(float *ip, int size ) {
    // generate different seed for random number.

    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i ++ ) {
        ip[i] = (float) (rand() & 0xff ) / 10.0f;
    }
}

double cpuSecond() {
    struct timeval tp; 
    gettimeofday(&tp, NULL);
    return (( double )tp.tv_sec + (double) tp.tv_usec * 1.e-6);

}
int main(int argc, char ** argv) {
    printf("%s Starting ...\n", argv[0]);

    // setup device.

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using device %d. %s\n", dev, deviceProp.name);

    // setup date size of vectors.

    int nElem = 1 << 24;

    printf("Vector size %d.\n", nElem);
    
    // malloc host memory

    size_t nBytes = nElem * sizeof(float);
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float*) malloc(nBytes);
    h_B = (float*) malloc(nBytes);
    hostRef = (float*) malloc(nBytes);
    gpuRef = (float*) malloc(nBytes);
    
    double iStart, iElaps;

    // initialize data at host side.

    iStart = cpuSecond();
    initialData (h_A, nElem);
    initialData (h_B, nElem);
    iElaps = cpuSecond() - iStart;

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add vector at host side for result checks.

    // malloc device global memory
    
    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    // transfer data from host to device

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // invoke kernel at host side.

    int iLen=1024;
    dim3 block(iLen);
    dim3 grid (nElem + block.x-1 / block.x);
    
    iStart = cpuSecond();
 
    sumArraysOnGPU <<< grid, block >>>(d_A, d_B, d_C, nElem);
    hipDeviceSynchronize();
    iElaps = cpuSecond()-iStart;
    printf("sumArrayOnGPU: <<<%d, %d>>> Time Elapsed %f sec\n", grid.x, block.x, iElaps);

    // copy kernel result back to host side.

    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    // free device global memory.

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return 0;
}
