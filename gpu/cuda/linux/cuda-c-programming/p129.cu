#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#define DEBUG 0
inline double seconds()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

// Recursive Implementation of Interleaved Pair Approach

int cpuRecursiveReduce(int *data, int const size)
{
    if (DEBUG == 1) 
        printf("cpuRecursiveReduce: size: %u.\n", size);

    // Stop condition.

    if (size == 1) return data[0];

    // Renew the stride.

    int const stride = size / 2;

    // In-place reduction.

    for (int i = 0; i < stride; i++)
    {
        data[i] += data[i + stride];
    }

    // call recursively

    return cpuRecursiveReduce(data, stride);
}

__global__ void gpuRecursiveReduce(int * g_idata, int *g_odata, unsigned int isize) {
    
    // set thread ID.

    unsigned int tid = threadIdx.x;

   // Convert global data pointer to the local pointer of this block.

    int * idata = g_idata + blockIdx.x * blockDim.x;
    int * odata = &g_odata[blockIdx.x];

    // Stop condition.

    if (isize == 2 && tid == 0) {
        g_odata[blockIdx.x] = idata[0] + idata[1];
        return ;
    }

    // Nested invocation.

    int istride = isize >> 1;
    if ( istride > 1 && tid < istride ) {
        // in place reduction.
        
        idata[tid] += idata[tid + istride];
    }

    // Sync at block level.

    __syncthreads();

    // Nested invocation to generate child grids.

    if(tid == 0) {
        gpuRecursiveReduce<<<1, istride>>>(idata, odata, istride);

        // Sync all child grids launched in this block.

        hipDeviceSynchronize();
    }

    // Sync at block level again.

    __syncthreads();
}

__global__ void reduceNeighbored(int * g_idata, int * g_odata, unsigned int n) {
    // Set thread iD.

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block.
    // Offset into current block from the beginning of data stream.

    int * idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check.

    if (idx >= n ) return ;

    // In-place reduction in global memory.
    // Stride is multiple for every loop until blockDim is reached.
    // if even number of threads, add current even value plus value at stride away from current even.

    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if ((tid % (2 * stride)) == 0 ) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within block.

        __syncthreads();
    }

    // Write result for this block to global mem.

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}


__global__ void warmup(int * g_idata, int * g_odata, unsigned int n) {
    // Set thread iD.

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block.
    // Offset into current block from the beginning of data stream.

    int * idata = g_idata + blockIdx.x + blockDim.x;

    // Boundary check.

    if (idx >= n ) return ;

    // In-place reduction in global memory.
    // Stride is multiple for every loop until blockDim is reached.
    // if even number of threads, add current even value plus value at stride away from current even.

    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if ((tid % (2 * stride)) == 0 ) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within block.

        __syncthreads();
    }

    // Write result for this block to global mem.

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}


int main(int argc, char ** argv) {
    // Setup device.

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev); 
    printf("%s starting reduction at ", argv[0]);
    printf("Device %d: %s.", dev, deviceProp.name);
    hipSetDevice(dev);

    bool bResult = false;
    
    // Initialization.

    int size = 1 << 24; // total No. of elements.
    printf(" with array size %d.", size);

    // Execution configuration.

    int blocksize = 512; // init. block size.
    if (argc > 1 ) {
        blocksize = atoi(argv[1]);
    }

    dim3 block (blocksize, 1);
    dim3 grid ((size+block.x-1)/block.x, 1);
    
    printf("grid %d block %d\n", grid.x, block.x);

    // Allocate host memory.

    size_t bytes = size * sizeof(int);
    int *h_idata = (int*) malloc(bytes);

    // h_odata will hold the sum of reduction in every block therefore this is 
    // a grid size or total No. of block in grid. 

    int *h_odata = (int*) malloc(grid.x * sizeof(int));
    int *tmp = (int*) malloc(bytes);

    // Initialize the array.

    for (int i = 0; i < size; i ++ ) {
        // mask off high 2 bytes to  force max number to 255.

        h_idata[i] = (int)(rand() & 0xff);
    }    
    memcpy(tmp, h_idata, bytes);

    size_t iStart, iElaps;
    int gpu_sum = 0;
    
    // Allocate device memory.

    int * d_idata = NULL;
    int * d_odata = NULL;
    hipMalloc((void **) &d_idata, bytes);
    hipMalloc((void **) &d_odata, grid.x * sizeof(int));

    // CPU reduction.  I am going to skip this one.
    
    iStart = seconds();
    int cpu_sum = cpuRecursiveReduce(tmp, size);
    iElaps = seconds() - iStart;

    printf("cpu reduce elapsed %d ms cpu_sum: %d\n", iElaps, cpu_sum);

    // kernel 1: reduceNeighbored.

    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = seconds();
    warmup<<<grid, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();
    iElaps = seconds() - iStart;
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) 
        gpu_sum += h_odata[i];

    //printf("GPU warmup elapsed %d ms gpu_sum: %d <<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x, block.x);
    printf("GPU warmup elapsed %d.\n", iElaps);

    // kernel 1: reduceNeighbored.

    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = seconds();
    reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();
    iElaps = seconds() - iStart;
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) 
        gpu_sum += h_odata[i];

    printf("GPU Neighbored elapsed %d ms gpu_sum: %d <<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x, block.x);

    // kernel 2: reduceNeighbored.

    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = seconds();
    gpuRecursiveReduce<<<grid, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();
    iElaps = seconds() - iStart;
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) 
        gpu_sum += h_odata[i];

    printf("GPU recursive reduce  %d ms gpu_sum: %d <<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x, block.x);


    // Free host memory.

    free(h_idata);
    free(h_odata);

    // Free device memory.

    hipFree(d_idata);
    hipFree(d_odata);

    // Reset device.

    hipDeviceReset();

}
