#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#define ENABLE_34 1
inline double seconds()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

__global__ void warmingUp(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if ((tid / warpSize) % 2 == 0) {
        ia = 100.0f;
    } else {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void mathKernel1(float * c) { 
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a = b = 0.0f;

    if (tid % 2 == 0) {
        a = 100.0f;        
    } else {
        b = 200.0f;
    }
    c[tid] = a + b;
}

__global__ void mathKernel2(float * c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float a, b;
    a = b = 0.0f;

    if ((tid / warpSize) % 2 == 0) {
        a = 100.0f;        
    } else {
        b = 200.0f;
    }
    c[tid] = a + b;
}

__global__ void mathKernel3(float * c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib, ic, id;
    ia = ib = ic = id = 0.0f;

    switch(tid % 8)  {
        case 0:
        case 4:
            ia = 100.0f;
        case 1:
        case 5:
            ia = 200.0f;
        case 2:
        case 6:
            ia = 300.0f;
        case 3:
        case 7:
            ia = 400.0f;
    }

    c[tid] = ia + ib + ic + id;
}

__global__ void mathKernel4(float *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    int itid = tid >> 5;

    if (itid & 0x01 == 0) {
        ia = 100.0f;
    } else {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

int main(int argc, char **argv) {
    // setup device.

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s using Device %d: %s.\n", argv[0], dev, deviceProp.name);
    hipSetDevice(dev);

    // setup data size.

    int size = 64;
    int blocksize = 64;
    if (argc > 1) blocksize = atoi(argv[1]);
    if (argc > 2) size = atoi(argv[2]);

    printf("blocksize / size %d, %d.\n", blocksize, size);

    // setup execution congiguration.

    dim3 block(blocksize, 1);
    dim3 grid((size + block.x-1 ) / block.x, 1);
    
    printf("Execution configure (block %d grid %d).\n", block.x, grid.x);
    sleep(3);
    
    // allocate gpu memory.

    float *d_C;
    size_t nBytes = size * sizeof(float);
    hipMalloc((float**)&d_C, nBytes);
    
    // run a warmup kernel to remove overhead.

    size_t iStart, iElaps;
    hipDeviceSynchronize();
    iStart = seconds();
    warmingUp<<<grid,block>>>(d_C);
    hipDeviceSynchronize();
    iElaps = seconds() - iStart;
    //printf("warmup <<< %d %d >>> elapsed %d sec.\n", grid.x, block.x, iElaps);

    // run kernel 1.
    // tid%2 causes even numbered threads to take "if" clause and off numbered threads to take "else" clause.
    // 

    iStart = seconds();
    mathKernel1<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    iElaps = seconds() - iStart;
    printf("mathKernel1 <<< %4d %4d >>> elapsed %d sec.\n", grid.x, block.x, iElaps);

    // run kernel 2.
    //(tid/warpsize)%2 == 0 test causes branch granularity to multiple of warpsize. 

    iStart = seconds();
    mathKernel2<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    iElaps = seconds() - iStart;
    printf("mathKernel2 <<< %4d %4d >>> elapsed %d sec.\n", grid.x, block.x, iElaps);

    if (ENABLE_34 == 1) {

        // run kernel 3.
    
        iStart = seconds();
        mathKernel3<<<grid, block>>>(d_C);
        hipDeviceSynchronize();
        iElaps = seconds() - iStart;
        printf("mathKernel3 <<< %4d %4d >>> elapsed %d sec.\n", grid.x, block.x, iElaps);
    
        // run kernel 4.
    
        iStart = seconds();
        mathKernel4<<<grid, block>>>(d_C);
        hipDeviceSynchronize();
        iElaps = seconds() - iStart;
        printf("mathKernel4 <<< %4d %4d >>> elapsed %d sec.\n", grid.x, block.x, iElaps);
	}
	
    hipFree(d_C);
    hipDeviceReset();
    return 0;
}
