#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

inline double seconds()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

__global__ void reduceNeighbored(int * g_idata, int * g_odata, unsigned int n) {
    // Set thread iD.

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block.
    // Offset into current block from the beginning of data stream.

    int * idata = g_idata + blockIdx.x + blockDim.x;

    // Boundary check.

    if (idx >= n ) return ;

    // In-place reduction in global memory.
    // Stride is multiple for every loop until blockDim is reached.
    // if even number of threads, add current even value plus value at stride away from current even.

    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if ((tid % (2 * stride)) == 0 ) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within block.

        __syncthreads();
    }

    // Write result for this block to global mem.

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}


__global__ void warmup(int * g_idata, int * g_odata, unsigned int n) {
    // Set thread iD.

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block.
    // Offset into current block from the beginning of data stream.

    int * idata = g_idata + blockIdx.x + blockDim.x;

    // Boundary check.

    if (idx >= n ) return ;

    // In-place reduction in global memory.
    // Stride is multiple for every loop until blockDim is reached.
    // if even number of threads, add current even value plus value at stride away from current even.

    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if ((tid % (2 * stride)) == 0 ) {
            idata[tid] += idata[tid + stride];
        }

        // synchronize within block.

        __syncthreads();
    }

    // Write result for this block to global mem.

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}


int main(int argc, char ** argv) {
    // Setup device.

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev); 
    printf("%s starting reduction at ", argv[0]);
    printf("Device %d: %s.", dev, deviceProp.name);
    hipSetDevice(dev);

    bool bResult = false;
    
    // Initialization.

    int size = 1 << 24; // total No. of elements.
    printf(" with array size %d.", size);

    // Execution configuration.

    int blocksize = 512; // init. block size.
    if (argc > 1 ) {
        blocksize = atoi(argv[1]);
    }

    dim3 block (blocksize, 1);
    dim3 grid ((size+block.x-1)/block.x, 1);
    
    printf("grid %d block %d\n", grid.x, block.x);

    // Allocate host memory.

    size_t bytes = size * sizeof(int);
    int *h_idata = (int*) malloc(bytes);

    // h_odata will hold the sum of reduction in every block therefore this is 
    // a grid size or total No. of block in grid. 

    int *h_odata = (int*) malloc(grid.x * sizeof(int));
    int *tmp = (int*) malloc(bytes);

    // Initialize the array.

    for (int i = 0; i < size; i ++ ) {
        // mask off high 2 bytes to  force max number to 255.

        h_idata[i] = (int)(rand() & 0xff);
    }    
    memcpy(tmp, h_idata, bytes);

    size_t iStart, iElaps;
    int gpu_sum = 0;
    
    // Allocate device memory.

    int * d_idata = NULL;
    int * d_odata = NULL;
    hipMalloc((void **) &d_idata, bytes);
    hipMalloc((void **) &d_odata, grid.x * sizeof(int));

    // CPU reduction.  I am going to skip this one.
    /*
    iStart = seconds();
    int cpu_sum = re
    */

    // kernel 1: reduceNeighbored.

    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = seconds();
    warmup<<<grid, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();
    iElaps = seconds() - iStart;
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) 
        gpu_sum += h_odata[i];

    printf("GPU warmup elapsed %d ms gpu_sum: %d <<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x, block.x);

    // kernel 1: reduceNeighbored.

    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = seconds();
    reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();
    iElaps = seconds() - iStart;
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) 
        gpu_sum += h_odata[i];

    printf("GPU Neighbored elapsed %d ms gpu_sum: %d <<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x, block.x);

    hipDeviceSynchronize();
    iElaps = seconds() - iStart;
    hipMemcpy(h_odata, d_odata, grid.x/8 * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;

    for (int i = 0; i < grid.x/8; i++) 
        gpu_sum += h_odata[i];

    printf("GPU Cmptnrollelapsed %d ms gpu_sum: %d <<<grid %d block %d>>>\n", iElaps, gpu_sum, grid.x/8, block.x);

    // Free host memory.

    free(h_idata);
    free(h_odata);

    // Free device memory.

    hipFree(d_idata);
    hipFree(d_odata);

    // Reset device.

    hipDeviceReset();

}
